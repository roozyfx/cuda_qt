#include "hip/hip_runtime.h"
#include "cudaImageFuncs.h"
#include <iostream>

const size_t CHANNELS = 3;

__global__ void colorToGrayScale(uchar* Pin, uchar* Pout, size_t width, size_t height)
{
    size_t row { blockIdx.y * blockDim.y + threadIdx.y };
    size_t col { blockIdx.x * blockDim.x + threadIdx.x };
    if (row < height && col < width) {
        size_t grayScaleOffset { row * width + col };
        size_t colorOffset { grayScaleOffset * CHANNELS };
        uchar r { Pin[colorOffset + 0] };
        uchar g { Pin[colorOffset + 1] };
        uchar b { Pin[colorOffset + 2] };

        Pout[grayScaleOffset] = r * 0.21f + g * 0.72f + b * 0.07f;
        // Pout[grayScaleOffset] = static_cast<uchar>(r * 0.21f + g * 0.72f + b * 0.07f);
    }
}
CudaImageFuncs::CudaImageFuncs()
{
    _imageResult = new QImage();
}

CudaImageFuncs::~CudaImageFuncs()
{
    std::cout << "CudaImageFuncs destructor" << std::endl;
    if (_image)
        delete _image;
    if (_imageResult)
        delete _imageResult;
    // Free the memory on device
    std::cout << "free cuda memory" << std::endl;
    hipFree(_inputImage_d);
    hipFree(_outputImage_d);
}

void CudaImageFuncs::openImage()
{
    QWidget* temp { new QWidget() };
    auto imageFile { QFileDialog::getOpenFileName(temp, tr("Open Image"), "/home/user/fx/cuda/img",
        tr("Image Files (*.png "
           "*.jpg *.jpeg *.JPG "
           "*.JPEG *.bmp)")) };
    _image = new QImage(imageFile);
    *_image = _image->convertToFormat(QImage::Format_RGB32);
    delete temp;
}

void CudaImageFuncs::grayScale(bool bReleased)
{
    if (bReleased) {
        std::cout << "Gray Scale Button\n";
        if (_image) {
            // FXTODO: check for the format. If color or already in grayscale, etc.
            size_t width { static_cast<size_t>(_image->width()) };
            size_t height { static_cast<size_t>(_image->height()) };
            size_t size { width * height * sizeof(uchar) };
            size_t size_color { size * CHANNELS };
            uchar* result_h = new uchar[size];

            std::cout << "Point A" << std::endl;
            // allocate memory on device
            hipMalloc((void**)&_inputImage_d, size * CHANNELS);
            hipMalloc((void**)&_outputImage_d, size);

            std::cout << "Point B" << std::endl;
            // Copy data from host to device
            hipMemcpy(_inputImage_d, _image->bits(), size_color, hipMemcpyHostToDevice);
            std::cout << "Point C" << std::endl;

            // FXTODO: Check if correct gridDim, blockDim set
            // dim3 gridDim(std::ceil(width / 32.f), std::ceil(height / 32.f), 1);
            // dim3 blockDim(32, 32, 1);
            dim3 blockDim(32, 32);
            dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
            colorToGrayScale<<<gridDim, blockDim>>>(_inputImage_d, _outputImage_d,
                width, height);

            std::cout << "Point D" << std::endl;
            // Copy the results to host
            hipMemcpy(result_h, _outputImage_d, size, hipMemcpyDeviceToHost);
            *_imageResult = QImage(result_h, width, height, QImage::Format_Grayscale8).copy();

            std::cout << "Point F" << std::endl;
            delete[] result_h;
            emit sigShowResult();
        }
    }
}

void CudaImageFuncs::blur(bool bReleased)
{
    if (bReleased) {
        std::cout << "Blur Button\n";
        if (!_imageResult) {
            _imageResult = new QImage();
        }
        *_imageResult = QImage(QString("/home/user/fx/cuda/img/Felixkula.jpeg"));
        emit sigShowResult();
    }
}

void CudaImageFuncs::reset(bool bReleased)
{
    if (bReleased) {
        std::cout << "Reset Button\n";
    }
}
